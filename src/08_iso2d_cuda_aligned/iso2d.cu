#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2012, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived 
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Paulius Micikevicius (pauliusm@nvidia.com)
 * Max Grossman (jmg3@rice.edu)
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include <unistd.h>

/*
 * Before tackling this TODO, please read the TODO lower in this file that talks
 * about modifications to dimx.
 *
 * Once you have tackled the below TODO, you will need to modify this macro so
 * that the zeroth element in each row starts at an even multiple of 128 bytes.
 * The POINT_OFFSET macro is used to compute the offset in the curr, next, vsq
 * arrays of an (x, y) coordinate. Please note that x and y may be negative when
 * passed to this macro.
 *
 * By default, it computes this by multiplying the offset in the y direction
 * ('radius' rows of padding + 'y' rows) by 'dimx', the size of each row. It
 * then adds an offset of 'radius' elements of padding + 'x' columns. However,
 * given x=0 for any y, the byte alignment is not guaranteed to be 128 bytes
 * even if rows are always a multiple of 128 bytes because of the radius offset.
 */
#define POINT_OFFSET(x, y, dimx, radius) \
    (((radius) + (y)) * (dimx) + ((radius) + (x)))

#include "common.h"
#include "common2d.h"

#define BDIMX   32
#define BDIMY   16

__constant__ TYPE const_c_coeff[NUM_COEFF];

__global__ void fwd_kernel(TYPE *next, TYPE *curr, TYPE *vsq, int nx, int ny,
        int dimx, int radius) {
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int this_offset = POINT_OFFSET(x, y, dimx, radius);

    TYPE div = const_c_coeff[0] * curr[this_offset];
    for (int d = 1; d <= radius; d++) {
        const int y_pos_offset = POINT_OFFSET(x, y + d, dimx, radius);
        const int y_neg_offset = POINT_OFFSET(x, y - d, dimx, radius);
        const int x_pos_offset = POINT_OFFSET(x + d, y, dimx, radius);
        const int x_neg_offset = POINT_OFFSET(x - d, y, dimx, radius);
        div += const_c_coeff[d] * (curr[y_pos_offset] +
                curr[y_neg_offset] + curr[x_pos_offset] +
                curr[x_neg_offset]);
    }

    const TYPE temp = 2.0f * curr[this_offset] - next[this_offset];
    next[this_offset] = temp + div * vsq[this_offset];
}

int main( int argc, char *argv[] ) {
    config conf;
    setup_config(&conf, argc, argv);
    init_progress(conf.progress_width, conf.nsteps, conf.progress_disabled);

    if (conf.nx % BDIMX != 0) {
        fprintf(stderr, "Invalid nx configuration, must be an even multiple of "
                "%d\n", BDIMX);
        return 1;
    }
    if (conf.ny % BDIMY != 0) {
        fprintf(stderr, "Invalid ny configuration, must be an even multiple of "
                "%d\n", BDIMY);
        return 1;
    }

    TYPE dx = 20.f;
    TYPE dt = 0.002f;

    /*
     * TODO Change dimx below to ensure that each row of curr, next, vsq starts
     * at a 128-byte aligned boundary.
     *
     * The fundamental problem with the current code is that if the following is
     * not guaranteed to be an even multiple of 128 bytes:
     *
     *     (conf.nx + 2 * conf.radius) * sizeof(TYPE)
     *
     * then all rows of the allocated 2D matrices other than the first are
     * likely to start on mis-aligned byte boundaries.
     *
     * Therefore, the key change to make is to modify dimx such that each row
     * starts at a 128-byte boundary (i.e. the size of each row is itself a
     * multiple of 128 bytes).
     */

    // compute the pitch for perfect coalescing
    size_t dimx = conf.nx + 2*conf.radius;
    size_t dimy = conf.ny + 2*conf.radius;
    size_t nbytes = dimx * dimy * sizeof(TYPE);

    if (conf.verbose) {
        printf("x = %zu, y = %zu\n", dimx, dimy);
        printf("nsteps = %d\n", conf.nsteps);
        printf("radius = %d\n", conf.radius);
    }

    TYPE c_coeff[NUM_COEFF];
    TYPE *curr = (TYPE *)malloc(nbytes);
    TYPE *next = (TYPE *)malloc(nbytes);
    TYPE *vsq  = (TYPE *)malloc(nbytes);
    if (curr == NULL || next == NULL || vsq == NULL) {
        fprintf(stderr, "Allocations failed\n");
        return 1;
    }

    config_sources(&conf.srcs, &conf.nsrcs, conf.nx, conf.ny, conf.nsteps);
    TYPE **srcs = sample_sources(conf.srcs, conf.nsrcs, conf.nsteps, dt);

    init_data(curr, next, vsq, c_coeff, dimx, dimy, dimx * sizeof(TYPE), dx, dt);

    TYPE *d_curr, *d_next, *d_vsq;
    CHECK(hipMalloc((void **)&d_curr, nbytes));
    CHECK(hipMalloc((void **)&d_next, nbytes));
    CHECK(hipMalloc((void **)&d_vsq, nbytes));

    dim3 block(BDIMX, BDIMY);
    dim3 grid(conf.nx / block.x, conf.ny / block.y);

    double mem_start = seconds();

    CHECK(hipMemcpy(d_curr, curr, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_next, next, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_vsq, vsq, nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(const_c_coeff), c_coeff, NUM_COEFF * sizeof(TYPE)));
    double start = seconds();
    for (int step = 0; step < conf.nsteps; step++) {
        for (int src = 0; src < conf.nsrcs; src++) {
            if (conf.srcs[src].t > step) continue;
            int src_offset = POINT_OFFSET(conf.srcs[src].x, conf.srcs[src].y,
                    dimx, conf.radius);
            CHECK(hipMemcpy(d_curr + src_offset, srcs[src] + step,
                        sizeof(TYPE), hipMemcpyHostToDevice));
        }

        fwd_kernel<<<grid, block>>>(d_next, d_curr, d_vsq, conf.nx, conf.ny,
                dimx, conf.radius);
        TYPE *tmp = d_next;
        d_next = d_curr;
        d_curr = tmp;

        update_progress(step + 1);
    }
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    double compute_s = seconds() - start;

    CHECK(hipMemcpy(curr, d_curr, nbytes, hipMemcpyDeviceToHost));
    double total_s = seconds() - mem_start;

    float point_rate = (float)conf.nx * conf.ny / (compute_s / conf.nsteps);
    printf("iso_r4_2x:   %8.10f s total, %8.10f s/step, %8.2f Mcells/s/step\n",
            total_s, compute_s / conf.nsteps, point_rate / 1000000.f);

    if (conf.save_text) {
        save_text(curr, dimx, dimy, conf.ny, conf.nx, "snap.text", conf.radius);
    }

    free(curr);
    free(next);
    free(vsq);
    for (int i = 0; i < conf.nsrcs; i++) {
        free(srcs[i]);
    }
    free(srcs);

    CHECK(hipFree(d_curr));
    CHECK(hipFree(d_next));
    CHECK(hipFree(d_vsq));

    return 0;
}
