#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

/*
 * A simple example of performing matrix-vector multiplication using the cuBLAS
 * library and some randomly generated inputs.
 */

/*
 * M = # of rows
 * N = # of columns
 */
int M = 1024;
int N = 1024;

/*
 * Generate a vector of length N with random single-precision floating-point
 * values between 0 and 100.
 */
void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);

    for (i = 0; i < N; i++)
    {
        int r = rand();
        double dr = (double)r;
        X[i] = (dr / rMax) * 100.0;
    }

    *outX = X;
}

/*
 * Verify that Y = M * X
 */
static void verify(float *A, float *X, float *Y, int M, int N) {
    for (int row = 0; row < M; row++) {
        float sum = 0.0f;
        for (int col = 0; col < N; col++) {
            sum += A[row * N + col] * X[col];
        }

        if (sum != Y[row]) {
            fprintf(stderr, "Value mismatch at Y[%d].\n", row);
            fprintf(stderr, "Expected = %f\n", sum);
            fprintf(stderr, "CUBLAS   = %f\n", Y[row]);
            exit(1);
        }
    }
}

/*
 * Generate a matrix with M rows and N columns in column-major order. The matrix
 * will be filled with random single-precision floating-point values between 0
 * and 100.
 */
void generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);

    // For each column
    for (j = 0; j < N; j++)
    {
        // For each row
        for (i = 0; i < M; i++)
        {
            double dr = (double)rand();
            A[j * M + i] = (dr / rMax) * 100.0;
        }
    }

    *outA = A;
}

int main(int argc, char **argv)
{
    int i;
    float *A, *dA;
    float *X, *dX;
    float *Y, *dY;
    float beta;
    float alpha;

    alpha = 3.0f;
    beta = 4.0f;

    // Generate inputs
    srand(9384);
    generate_random_dense_matrix(M, N, &A);
    generate_random_vector(N, &X);
    Y = (float *)malloc(sizeof(float) * M);

    // TODO 1. Declare and create a CUBLAS handle using hipblasCreate

    /*
     * TODO 2. Allocate a float array on the device with M x N elements, to
     * store the matrix 'A' (hipMalloc).
     */

    /*
     * TODO 3. Allocate a float array on the device with N elements, to
     * store the input vector 'X' (hipMalloc).
     */

    /*
     * TODO 4. Allocate a float array on the device with M elements, to
     * store the output vector 'Y' (hipMalloc).
     */

    /*
     * TODO 5. Copy the input M x N matrix 'A' to the space you've allocated for
     * it on the device, using hipblasSetMatrix.
     */

    /*
     * TODO 6. Copy the input N-element vector 'X' to the space you've allocated
     * for it on the device, using hipblasSetVector.
     */

    /*
     * TODO 7. Call hipblasSgemv to perform the dense matrix-vector multiplication
     * M*X=Y. You may assume:
     *
     *   1. We do not wish to do a transpose.
     *   2. alpha = 3.0f (defined above).
     *   3. beta = 4.0f (defined above).
     */

    /*
     * TODO 8. Copy the output M-element vector 'Y' out of the CUDA device and
     * into the host buffer 'Y', using hipblasGetVector.
     */

    for (i = 0; i < 10; i++)
    {
        printf("%2.2f\n", Y[i]);
    }

    printf("...\n");

    verify(A, X, Y, M, N);

    free(A);
    free(X);
    free(Y);

    /*
     * TODO 9. Free any device arrays you have allocated, using hipFree.
     * Release the CUBLAS handle created, using hipblasDestroy.
     */

    return 0;
}
