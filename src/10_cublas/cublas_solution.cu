#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define CHECK_CUDA(call) { \
    const hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA ERROR @ %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err)); \
        exit(1); \
    } \
}

#define CHECK_CUBLAS(call)                                                     \
{                                                                              \
    hipblasStatus_t err;                                                        \
    if ((err = (call)) != HIPBLAS_STATUS_SUCCESS)                               \
    {                                                                          \
        fprintf(stderr, "Got CUBLAS error %d at %s:%d\n", err, __FILE__,       \
                __LINE__);                                                     \
        exit(1);                                                               \
    }                                                                          \
}

/*
 * A simple example of performing matrix-vector multiplication using the cuBLAS
 * library and some randomly generated inputs.
 */

/*
 * M = # of rows
 * N = # of columns
 */
int M = 1024;
int N = 1024;

/*
 * Generate a vector of length N with random single-precision floating-point
 * values between 0 and 100.
 */
void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);

    for (i = 0; i < N; i++)
    {
        int r = rand();
        double dr = (double)r;
        X[i] = (dr / rMax) * 10.0;
    }

    *outX = X;
}

/*
 * Verify that Y = M * X
 */
static void verify(float *A, float *X, float *Y, int M, int N, float alpha) {
    double avg_perc_err = 0.0;
    for (int row = 0; row < M; row++) {
        float sum = 0.0f;
        for (int col = 0; col < N; col++) {
            sum += alpha * A[row * N + col] * X[col];
        }

        avg_perc_err += fabs(Y[row] - sum) / sum;
    }
    avg_perc_err /= (float)M;
    printf("\n%% error = %f%%\n", 100.0 * avg_perc_err);
}

/*
 * Generate a matrix with M rows and N columns in column-major order. The matrix
 * will be filled with random single-precision floating-point values between 0
 * and 100.
 */
void generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);

    // For each column
    for (j = 0; j < N; j++)
    {
        // For each row
        for (i = 0; i < M; i++)
        {
            double dr = (double)rand();
            A[j * M + i] = (dr / rMax) * 100.0;
        }
    }

    *outA = A;
}

int main(int argc, char **argv)
{
    int i;
    float *A, *dA;
    float *X, *dX;
    float *Y, *dY;
    float beta;
    float alpha;
    hipblasHandle_t handle = 0;

    alpha = 3.0f;
    beta = 4.0f;

    // Generate inputs
    srand(9384);
    generate_random_dense_matrix(M, N, &A);
    generate_random_vector(N, &X);
    Y = (float *)malloc(sizeof(float) * M);
    memset(Y, 0x00, sizeof(float) * M);

    // Create the cuBLAS handle
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Allocate device memory
    CHECK_CUDA(hipMalloc((void **)&dA, sizeof(float) * M * N));
    CHECK_CUDA(hipMalloc((void **)&dX, sizeof(float) * N));
    CHECK_CUDA(hipMalloc((void **)&dY, sizeof(float) * M));

    // Transfer inputs to the device
    CHECK_CUBLAS(hipblasSetVector(N, sizeof(float), X, 1, dX, 1));
    CHECK_CUBLAS(hipblasSetVector(M, sizeof(float), Y, 1, dY, 1));
    CHECK_CUBLAS(hipblasSetMatrix(M, N, sizeof(float), A, M, dA, M));

    // Execute the matrix-vector multiplication
    CHECK_CUBLAS(hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, dA, M, dX, 1,
                             &beta, dY, 1));

    // Retrieve the output vector from the device
    CHECK_CUBLAS(hipblasGetVector(M, sizeof(float), dY, 1, Y, 1));

    for (i = 0; i < 10; i++)
    {
        printf("%2.2f\n", Y[i]);
    }

    printf("...\n");

    verify(A, X, Y, M, N, alpha);

    free(A);
    free(X);
    free(Y);

    CHECK_CUDA(hipFree(dA));
    CHECK_CUDA(hipFree(dY));
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}
